#include "hip/hip_runtime.h"
#include <iostream>
#include <string>

#include <hipfft/hipfft.h>

#include "CLI11.hpp"

#include "cudaAtScaleFinalAssignment/ImageProcessor.hpp"
#include "cudaAtScaleFinalAssignment/PgmDataGetter.hpp"




#include "hip/hip_runtime.h"
#include ""
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

#include <ctime>
#include <time.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <hipfft/hipfft.h>
#include <fstream>

using namespace std;
typedef float2 Complex; // float2 is a two component vector that comes with cuda - https://stackoverflow.com/questions/4079451/what-about-the-types-int2-int3-float2-float3-etc


__global__ void ComplexMUL(Complex *a, Complex *b, unsigned int numElements)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < numElements) {
        a[i].x = a[i].x * b[i].x - a[i].y*b[i].y;
        a[i].y = a[i].x * b[i].y + a[i].y*b[i].x;
    }
}

int main(int argc, char** argv) {
    
    int N = 5;
    int SIZE = N*N;

    // create data on host with only x as 1 - x equals Re{fg[i]} and y equals Im{fg[i]} and fg is a row major matrice
    Complex *fg = new Complex[SIZE];
    for (int i = 0; i < SIZE; i++){
        fg[i].x = 1;
        fg[i].y = 0;
    }
    Complex *fig = new Complex[SIZE];
    for (int i = 0; i < SIZE; i++){
        fig[i].x = 1;
        fig[i].y = 0;
    }
    for (int i = 0; i < N * N; i = i + N)
    {
        for (int j=0; j < N; j++){
            cout << fg[i+j].x << " ";
        }
        cout << endl;
    }
    cout << "----------------" << endl;
    for (int i = 0; i < N * N; i = i + N)
    {
        for (int j=0; j < N; j++){
            cout << fig[i+j].x << " ";
        }
        cout << endl;
    }
    cout << "----------------" << endl;

    int mem_size = sizeof(Complex)* SIZE;

    // allocate memory on device that corresponds to the data on the host (CPU)
    hipfftComplex *d_signal;
    checkCudaErrors(hipMalloc((void **) &d_signal, mem_size)); 
    checkCudaErrors(hipMemcpy(d_signal, fg, mem_size, hipMemcpyHostToDevice));
    // same for the filter kernel
    hipfftComplex *d_filter_kernel;
    checkCudaErrors(hipMalloc((void **)&d_filter_kernel, mem_size));
    checkCudaErrors(hipMemcpy(d_filter_kernel, fig, mem_size, hipMemcpyHostToDevice));

    // cout << d_signal[1].x << endl;
    // CUFFT plan
    hipfftHandle plan;
    hipfftPlan2d(&plan, N, N, HIPFFT_C2C);

    // Transform signal and filter
    printf("Transforming signal hipfftExecR2C\n");
    hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD);
    hipfftExecC2C(plan, (hipfftComplex *)d_filter_kernel, (hipfftComplex *)d_filter_kernel, HIPFFT_FORWARD);

    printf("Launching Complex multiplication<<< >>>\n");
    ComplexMUL <<< N, N >> >(d_signal, d_filter_kernel, static_cast<unsigned int>(SIZE));

    // Transform signal back
    printf("Transforming signal back hipfftExecC2C\n");
    hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_BACKWARD);

    // create memory for the result on the CPU
    Complex *result = new Complex[SIZE];
    // copy back the result and print it to the terminal
    hipMemcpy(result, d_signal, sizeof(Complex)*SIZE, hipMemcpyDeviceToHost);
    for (int i = 0; i < SIZE; i = i + N)
    {
        for (int j=0; j < N; j++){
            cout << result[i+j].x << " ";
        }
        cout << endl;
    }

    delete result, fg, fig;
    hipfftDestroy(plan); // destroy the cufft plan
    //hipfftDestroy(plan2);
    // free the memory on GPU
    hipFree(d_signal);
    hipFree(d_filter_kernel);


    /*
    // command line parsing
    CLI::App app{"App description"};
    argv = app.ensure_utf8(argv);
    std::string pathToData = "default";
    std::string pathToOutput = "./output";
    app.add_option("-p,--path", pathToData, "Path to the tiff data. We only want the path. The program will iterate over all *.tiff data within this folder (in a non-recursive manner!).");
    app.add_option("-o,--output", pathToOutput, "Output folder where the transformed images should be stored.");
    CLI11_PARSE(app, argc, argv);



    
    // data input
    PgmDataGetter dataLoader(pathToData);

    // image processor encapsulation
    ImageProcessor imgProcessor(pathToOutput);

    // main loop
    std::cout << "Processing " << dataLoader.getNumImages() << " images" << std::endl;
    bool terminate = false;
    while (!terminate) {
        std::string tmpImgPath = dataLoader.getNextImage();
        if ("" == tmpImgPath || "Error" == tmpImgPath) {
            terminate = true;
            continue;
        }

        if (!imgProcessor.processImage(tmpImgPath)) {
            std::cerr << "Could not process image " << tmpImgPath << " properly." << std::endl;
        }
    }
    std::cout << "All images were processing. Terminating successfully." << std::endl;
    return EXIT_SUCCESS;
    */
}